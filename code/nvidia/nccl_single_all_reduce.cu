#include <iomanip>
#include <chrono>
#include <sstream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <thrust/device_ptr.h>
#include <thrust/fill.h>

#include "nccl_helper.h"
#include "cuda_helper.h"


int all_reduce(int t_size, hipStream_t * streams, ncclComm_t * comms, int numGpus, int numRepeats) {

    float ** send_buff = (float **)malloc(numGpus * sizeof(float *));
    float ** recv_buff = (float **)malloc(numGpus * sizeof(float *));


    for (int i = 0; i < numGpus; i++) {
        CHECK_CUDA_ERROR(hipSetDevice(i));
        CHECK_CUDA_ERROR(hipMalloc(send_buff+i, t_size * sizeof(float)));
        CHECK_CUDA_ERROR(hipMalloc(recv_buff+i, t_size * sizeof(float)));

        thrust::fill(thrust::device_ptr<float>(send_buff[i]),
                     thrust::device_ptr<float>(send_buff[i] + t_size), i);
        thrust::fill(thrust::device_ptr<float>(recv_buff[i]),
                     thrust::device_ptr<float>(recv_buff[i] + t_size), 0.f);

    }

    for (int i = 0; i < numGpus; i++) {
        CHECK_CUDA_ERROR(hipSetDevice(i));
        hipStreamSynchronize(streams[i]);
    }

    auto start = std::chrono::steady_clock::now();

    for (int i = 0; i < numRepeats; i++) {
        for (int i = 0; i < numGpus; i++) {
            CHECK_CUDA_ERROR(hipSetDevice(i));
            CHECK_NCCL_ERROR(ncclAllReduce((void *) (send_buff[i]),
                                           (void *) (recv_buff[i]),
                                           t_size,
                                           ncclFloat,
                                           ncclSum,
                                           comms[i],
                                           streams[i]), 0);
        }

        for (int i = 0; i < numGpus; i++) {
            CHECK_CUDA_ERROR(hipSetDevice(i));
            hipStreamSynchronize(streams[i]);
        }
    }

    auto end = std::chrono::steady_clock::now();
    int time = static_cast<int>(std::chrono::duration<double, std::micro>(end - start).count() / numRepeats);


    for (int i = 0; i < numGpus; i++) {
        hipFree(send_buff[i]);
        hipFree(recv_buff[i]);
    }

    free(send_buff);
    free(recv_buff);

    return time;

}


int main(int argc, char  **argv) {

    hipFree(0);
    int nVis;

    int numRepeats = 1000;
    int numGpus;

    CHECK_CUDA_ERROR(hipGetDeviceCount(&nVis));

    if (argc > 1) {
        numGpus = atoi(argv[1]);
    } else {
        throw std::runtime_error("Must specify number of GPUs!");
    }

    if (argc > 2) {
        numRepeats = atoi(argv[2]);
    }

    if (numGpus > nVis) {
        std::stringstream ss;
        ss << "Number of Gpus Requested: " << numGpus << std::endl;
        ss << "Number of devices visible: " << nVis << std::endl;
        ss << "Number of Gpus requested cannot be more than visible devices" << std::endl;
        throw std::runtime_error(ss.str());
    }

    // Initialize curand_gen and set appropriate seed.
    hiprandGenerator_t curand_gen;
    hiprandCreateGenerator(&curand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(curand_gen, 123ULL);

    std::vector<int> devList;
    for (int i = 0; i < numGpus; i++) 
        devList.push_back(i);

    ncclComm_t* comms = (ncclComm_t*)malloc(sizeof(ncclComm_t)*numGpus);
    CHECK_NCCL_ERROR(ncclCommInitAll(comms, numGpus, devList.data()), 0);

    hipStream_t * streams = (hipStream_t*)malloc(sizeof(hipStream_t)*numGpus);
    for (int i = 0; i < numGpus; i++) {
        CHECK_CUDA_ERROR(hipSetDevice(i));
        CHECK_CUDA_ERROR(hipStreamCreate(streams+i));
    }

    std::cout << " NCCL AllReduce " << std::endl;
    std::cout << " Num Ranks: " << numGpus << std::endl;

    std::cout << std::setfill('-') << std::setw(75) << "-" << std::endl;
    std::cout << std::setfill(' ');
    std::cout << "    # of floats    bytes transferred    Time (msec)   " << std::endl;

    std::cout << std::setfill('-') << std::setw(75) << "-" << std::endl;
    std::cout << std::setfill(' ');


    std::vector<int> sizes = {100000, 3097600, 4194304, 6553600, 16777217};


    for (auto t_size: sizes) {
        int time  = all_reduce(t_size, streams, comms, numGpus, numRepeats);
        float time_ms = time/1000.0;
        std::cout << std::setw(15) << t_size << std::setw(15) << t_size * 4 << std::setw(20) << time_ms << std::endl;
    }

    for (int i = 0; i < numGpus; i++) {
        ncclCommDestroy(comms[i]);
    }

    free(streams);
    free(comms);

}
